#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optix.h>
#include "opg/raytracing/optixglm.h"

#include "raycastingraygenerator.cuh"

#include "opg/scene/utility/interaction.cuh"
#include "opg/scene/utility/trace.cuh"
#include "opg/hostdevice/color.h"

__constant__ RayCastingLaunchParams params;

extern "C" __global__ void __miss__main()
{
    SurfaceInteraction *si = getPayloadDataPointer<SurfaceInteraction>();

    const glm::vec3 world_ray_origin = optixGetWorldRayOriginGLM();
    const glm::vec3 world_ray_dir    = optixGetWorldRayDirectionGLM();
    const float     tmax             = optixGetRayTmax();

    si->incoming_ray_dir = world_ray_dir;

    // No valid interaction found, set incoming_distance to NaN
    si->set_invalid();
}

extern "C" __global__ void __raygen__main()
{
    const glm::uvec3 launch_idx  = optixGetLaunchIndexGLM();
    const glm::uvec3 launch_dims = optixGetLaunchDimensionsGLM();

    // Index of current pixel in image
    const glm::uvec2 pixel_index = glm::uvec2(launch_idx.x, launch_idx.y);

    /* Implement:
     * - Generate camera rays
     */
    glm::vec3 ray_origin = params.camera.eye;

    float u = ((pixel_index.x + 0.5) - (image_width/2)) / (image_width/2);
    float v = ((pixel_index.y + 0.5) - (image_height/2)) / (image_height/2);
    glm::vec3 ray_dir = glm::normalize(params.camera.W + u * params.camera.U + v * params.camera.V);
    //

    SurfaceInteraction si;
    traceWithDataPointer<SurfaceInteraction>(
            params.traversable_handle,
            ray_origin,
            ray_dir,
            0.0f,                                   // tmin: Start ray at ray_origin + tmin * ray_direction
            std::numeric_limits<float>::infinity(), // tmax: End ray at ray_origin + tmax * ray_direction
            params.traceParams,
            &si
    );

    glm::vec3 result = si.normal * 0.5f + 0.5f;
    if (!si.is_finite()) result = glm::vec3(0);

    // Write linear output color, want to interpret the result as "perceptually" linear colors with sRGB gamma applied.
    params.output_buffer(pixel_index).value() = apply_inverse_srgb_gamma(glm::clamp( result, 0.0f, 1.0f ));
}
