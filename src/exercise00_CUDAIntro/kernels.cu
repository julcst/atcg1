#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "opg/hostdevice/random.h"
#include "opg/glmwrapper.h"
#include "opg/hostdevice/misc.h"
#include "opg/exception.h"
#include <cstdint>
#include <cstdio>

#include "kernels.h"

// By default, .cu files are compiled into .ptx files in our framework, that are then loaded by OptiX and compiled
// into a ray-tracing pipeline. In this case, we want the kernels.cu to be compiled as a "normal" .obj file that is
// linked against the application such that we can simply call the functions defined in the kernels.cu file.
// The following custom pragma notifies our build system that this file should be compiled into a "normal" .obj file.
#pragma cuda_source_property_format = OBJ

__global__ void vecMulConst(int* dataArray, int N, int constant)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        dataArray[i] *= constant;
}

void launchVecMulConst(int* dataArray, int N, int constant){
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vecMulConst<<<blocksPerGrid, threadsPerBlock>>>(dataArray, N, constant);
    CUDA_SYNC_CHECK(); 
}
 
__global__ void convolution2D(const unsigned char* image, const int* kernel, int* output, int image_width, int image_height, int kernel_size)
{
    //pixel index of flattened image
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < image_width * image_height) {
        int kernel_radius = kernel_size / 2;
        int row = index / image_width;
        int col = index % image_width;
        int sum = 0;
        for (int ky = -kernel_radius; ky <= kernel_radius; ++ky) {
            for (int kx = -kernel_radius; kx <= kernel_radius; ++kx) {
                //index of neighbor pixel
                int n_row = min(max(row + ky, 0), image_height - 1);
                int n_col = min(max(col + kx, 0), image_width - 1);
                int n_index = n_row * image_width + n_col;

                //perform convolution
                sum += image[n_index] * kernel[(ky + kernel_radius) * kernel_size + (kx + kernel_radius)];
            }
        }

        output[index] = sum;
    }
}

void launchConvolution2D(const unsigned char* image, const int* kernel, int* output, int image_width, int image_height, int kernel_size){
    int threadsPerBlock = 256;
    int blocksPerGrid = (image_width * image_height + threadsPerBlock - 1) / threadsPerBlock;
    convolution2D<<<blocksPerGrid, threadsPerBlock>>>(image, kernel, output, image_width, image_height, kernel_size);
    CUDA_SYNC_CHECK(); 
}

__global__ void generateRandom(float* out, uint32_t n) {
    // Get the thread index
    const auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= n) return; // Check if the thread index is within bounds

    // Generate a seed and init RNG
    const auto tea = sampleTEA32(idx, 42);
    PCG32 rng{};
    rng.seed(tea, 0);

    // Generate a random float in the range [0, 1)
    out[idx] = rng.nextFloat();
}

__global__ void countThreshold(const float* in, uint32_t* count, float threshold, uint32_t n) {
    // Get the thread index
    const auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= n) return; // Check if the thread index is within bounds

    // Count the number of elements above the threshold
    if (in[idx] > threshold) atomicAdd(count, 1);
}

void sampleBernoulli(uint32_t* out, float threshold, uint32_t n) {
    uint32_t* aboveThresholdCount; // compute this value
    hipMallocManaged(&aboveThresholdCount, sizeof(int));

    float* randoms;
    hipMallocManaged(&randoms, n * sizeof(float));

    uint32_t blockSize = 256;
    uint32_t blockCount = (n + blockSize - 1) / blockSize;

    generateRandom<<<blockCount, blockSize>>>(randoms, n);
    CUDA_SYNC_CHECK();
    countThreshold<<<blockCount, blockSize>>>(randoms, aboveThresholdCount, threshold, n);
    CUDA_SYNC_CHECK();

    *out = *aboveThresholdCount; // Copy the result to the output variable

    hipFree(randoms);
    hipFree(aboveThresholdCount);
}

__global__ void matMult(const float* A, int A_rows, int A_cols,
                        const float* B, int B_rows, int B_cols,
                        float* C, int C_rows, int C_cols) {
    // Get the thread index
    const auto row = blockIdx.y * blockDim.y + threadIdx.y;
    const auto col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < C_rows && col < C_cols) {
        float sum = 0.0f;
        for (int k = 0; k < A_cols; k++) {
            sum += A[row * A_cols + k] * B[k * B_cols + col];
        }
        C[row * C_cols + col] = sum;
    }
}

void matrixMultiply(const float* A, int A_rows, int A_cols,
                    const float* B, int B_rows, int B_cols,
                    float* C, int C_rows, int C_cols) {
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, A_rows * A_cols * sizeof(float));
    hipMemcpy(d_A, A, A_rows * A_cols * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&d_B, B_rows * B_cols * sizeof(float));
    hipMemcpy(d_B, B, B_rows * B_cols * sizeof(float), hipMemcpyHostToDevice);
    hipMallocManaged(&d_C, C_rows * C_cols * sizeof(float));

    dim3 blockSize(16, 16);
    dim3 gridSize((C_cols + blockSize.x - 1) / blockSize.x, (C_rows + blockSize.y - 1) / blockSize.y);

    matMult<<<gridSize, blockSize>>>(d_A, A_rows, A_cols, d_B, B_rows, B_cols, d_C, C_rows, C_cols);
    CUDA_SYNC_CHECK();
    
    hipMemcpy(C, d_C, C_rows * C_cols * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}