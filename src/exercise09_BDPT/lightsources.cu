#include "hip/hip_runtime.h"
#include "lightsources.cuh"

#include "opg/scene/interface/emitter.cuh"
#include "opg/scene/utility/interaction.cuh"
#include "opg/hostdevice/binarysearch.h"
#include "opg/hostdevice/coordinates.h"
#include "common.h"

#include <optix.h>


extern "C" __device__ EmitterSamplingResult __direct_callable__pointlight_sampleLight(const Interaction &si, PCG32 &unused_rng)
{
    const PointLightData *sbt_data = *reinterpret_cast<const PointLightData **>(optixGetSbtDataPointer());

    glm::vec3 dir_to_light = sbt_data->position - si.position;

    EmitterSamplingResult result;
    result.radiance_weight_at_receiver = sbt_data->intensity / glm::dot(dir_to_light, dir_to_light);
    result.direction_to_light = glm::normalize(dir_to_light);
    result.distance_to_light = glm::length(dir_to_light);
    result.normal_at_light = glm::vec3(0); // No surface normal!
    result.sampling_pdf = 1;
    return result;
}

extern "C" __device__ EmitterPhotonSamplingResult __direct_callable__pointlight_samplePhoton(PCG32 &rng)
{
    const PointLightData *sbt_data = *reinterpret_cast<const PointLightData **>(optixGetSbtDataPointer());

    EmitterPhotonSamplingResult result;
    result.position = sbt_data->position;
    result.direction = warp_square_to_sphere_uniform(rng.next2d());
    result.normal_at_light = glm::vec3(0); // No surface normal!
    result.sampling_pdf = warp_square_to_sphere_uniform_pdf(result.direction);
    result.radiance_weight = sbt_data->intensity / result.sampling_pdf;
    return result;
}


extern "C" __device__ EmitterSamplingResult __direct_callable__directionallight_sampleLight(const Interaction &si, PCG32 &unused_rng)
{
    const DirectionalLightData *sbt_data = *reinterpret_cast<const DirectionalLightData **>(optixGetSbtDataPointer());

    glm::vec3 dir_to_light = sbt_data->direction;

    EmitterSamplingResult result;
    result.radiance_weight_at_receiver = sbt_data->irradiance_at_receiver;
    result.direction_to_light = glm::normalize(dir_to_light);
    result.distance_to_light = std::numeric_limits<float>::infinity();
    result.normal_at_light = glm::vec3(0); // No surface normal!
    result.sampling_pdf = 1;
    return result;
}

extern "C" __device__ EmitterPhotonSamplingResult __direct_callable__directionallight_samplePhoton(PCG32 &rng)
{
    // Photon sampling for directional light not implemented
    // We would need to know a bounding volume of the scene, and then sample a position uniformly distributed on the bounding volume projected orthogonal to the light direction...
    EmitterPhotonSamplingResult result;
    result.radiance_weight = glm::vec3(0);
    result.sampling_pdf = 0; // invalid sample
    return result;
}






extern "C" __device__ glm::vec3 __direct_callable__spherelight_evalLight(const SurfaceInteraction &si)
{
    const SphereLightData *sbt_data = *reinterpret_cast<const SphereLightData **>(optixGetSbtDataPointer());
    // We can assume that si is actually on the surface of the light source

    // The emitted radiance is constant across the light source
    return sbt_data->radiance;
}

extern "C" __device__ EmitterSamplingResult __direct_callable__spherelight_sampleLight(const Interaction &si, PCG32 &rng)
{
    const SphereLightData *sbt_data = *reinterpret_cast<const SphereLightData **>(optixGetSbtDataPointer());

    // Some useful quantities
    glm::vec3 light_center_dir = glm::normalize(sbt_data->position - si.position);
    float light_center_distance = glm::length(sbt_data->position - si.position);
    float cap_height = 1 - glm::sqrt(light_center_distance*light_center_distance - sbt_data->radius*sbt_data->radius) / light_center_distance;

    // A transformation matrix that rotates the z axis to the light_center_dir
    glm::mat3 local_frame = opg::compute_local_frame(light_center_dir);


    EmitterSamplingResult result;

    if (light_center_distance < sbt_data->radius)
    {
        result.radiance_weight_at_receiver = glm::vec3(0);
        result.sampling_pdf = 0;
        return result;
    }

    // Light source sampling
    glm::vec3 local_direction_to_light = warp_square_to_spherical_cap_uniform(rng.next2d(), cap_height);
    result.direction_to_light = local_frame * local_direction_to_light;

    float spherical_cap_area = 2*glm::pi<float>() * cap_height;

    // Radiance divided by sampling pdf
    result.radiance_weight_at_receiver = sbt_data->radiance * spherical_cap_area;

    // Probability of sampling this direction via light source sampling
    result.sampling_pdf = 1 / spherical_cap_area;


    // Compute light_ray_length for occlusion query
    {
        glm::vec3 O = si.position - sbt_data->position;
        glm::vec3 D = result.direction_to_light;

        float p = glm::dot(D, O) / glm::dot(D, D); // p/2 actually
        float q = (glm::dot(O, O) - sbt_data->radius * sbt_data->radius) / glm::dot(D, D);

        float k = glm::max(p*p - q, 0.0f);
        // Usually we would have to check for k < 0, but by construction the ray intersects the light source.

        // Assuming the surface element is outside of the sphere, the first intersection is what we want!
        result.distance_to_light = -p - glm::sqrt(k);

        // Compute light source normal vector at position sampled on the light source surface.
        result.normal_at_light = glm::normalize(O + D * result.distance_to_light);
    }

    return result;
}

extern "C" __device__ float __direct_callable__spherelight_evalLightSamplingPDF(const Interaction &si, const SurfaceInteraction &si_on_light)
{
    const SphereLightData *sbt_data = *reinterpret_cast<const SphereLightData **>(optixGetSbtDataPointer());
    // We can assume that outgoing ray dir actually intersects the light source.

    // Some useful quantities
    float light_center_distance = glm::length(sbt_data->position - si.position);
    float cap_height = 1 - glm::sqrt(light_center_distance*light_center_distance - sbt_data->radius*sbt_data->radius) / light_center_distance;

    // Probability of sampling this direction via light source sampling
    return 1 / (2*M_PIf * cap_height);
}

extern "C" __device__ EmitterPhotonSamplingResult __direct_callable__spherelight_samplePhoton(PCG32 &rng)
{
    const SphereLightData *sbt_data = *reinterpret_cast<const SphereLightData **>(optixGetSbtDataPointer());

    EmitterPhotonSamplingResult result;
    result.sampling_pdf = 0; // invalid photon

    glm::vec3 light_normal = warp_square_to_sphere_uniform(rng.next2d());
    float light_normal_pdf = warp_square_to_sphere_uniform_pdf(light_normal);

    glm::mat3 light_normal_frame = opg::compute_local_frame(light_normal);
    glm::vec3 local_photon_direction = warp_square_to_hemisphere_cosine(rng.next2d());
    glm::vec3 photon_direction = light_normal_frame * local_photon_direction;
    float photon_dir_pdf = warp_square_to_hemisphere_cosine_pdf(local_photon_direction);

    glm::vec3 photon_origin = sbt_data->position + sbt_data->radius * light_normal;
    float photon_origin_pdf = light_normal_pdf / (sbt_data->radius * sbt_data->radius);

    // photon_weight = NdotL * sbt_data->radiance / (photon_origin_pdf * photon_dir_pdf);

    float NdotL = glm::dot(light_normal, photon_direction);

    // Populate result
    result.position = photon_origin;
    result.direction = photon_direction;
    result.normal_at_light = light_normal;

    // NdotL factor is due to reprojection of radiance from light surface to photon direction.
    // If NdotL close to 0 the sampled surface patch that emits a constant radiance covers only a smaller area when projected to the photon/light direction.
    //result.radiance = NdotL * sbt_data->radiance;
    float one_over_photon_origin_pdf = 4*glm::pi<float>()*sbt_data->radius*sbt_data->radius;
    //float one_over_photon_dir_pdf = glm::pi<float>()/NdotL;
    // NdotL cancels out:
    result.radiance_weight = sbt_data->radiance * glm::pi<float>() * one_over_photon_origin_pdf;
    result.sampling_pdf = photon_origin_pdf * photon_dir_pdf;

    return result;
}



extern "C" __device__ glm::vec3 __direct_callable__meshlight_evalLight(const SurfaceInteraction &si)
{
    const MeshLightData *sbt_data = *reinterpret_cast<const MeshLightData **>(optixGetSbtDataPointer());
    // We can assume that si is actually on the surface of the light source

    // The emitted radiance is constant across the light source
    return sbt_data->radiance;
}

extern "C" __device__ EmitterSamplingResult __direct_callable__meshlight_sampleLight(const Interaction &si, PCG32 &rng)
{
    const MeshLightData *sbt_data = *reinterpret_cast<const MeshLightData **>(optixGetSbtDataPointer());

    // Select the triangle to sample a direction from uniformly at random, proportional to its surface area
    uint32_t triangle_index = 0;
    // Sample the barycentric coordinates on the triangle uniformly.
    glm::vec2 triangle_barys = glm::vec2(0, 0);

    triangle_index = opg::binary_search(sbt_data->mesh_cdf, rng.next1d());

    triangle_barys = rng.next2d();
    // Mirror barys at diagonal line to cover a triangle instead of a square
    if (triangle_barys.x + triangle_barys.y > 1)
        triangle_barys = glm::vec2(1) - triangle_barys;


    // Compute the `light_position` using the triangle_index and the triangle_barys on the mesh:

    // Indices of triangle vertices in the mesh
    glm::uvec3 vertex_indices = glm::uvec3(0u);
    if (sbt_data->mesh_indices.elmt_byte_size == sizeof(glm::u32vec3))
    {
        // Indices stored as 32-bit unsigned integers
        const glm::u32vec3* indices = reinterpret_cast<glm::u32vec3*>(sbt_data->mesh_indices.data);
        vertex_indices = glm::uvec3(indices[triangle_index]);
    }
    else
    {
        // Indices stored as 16-bit unsigned integers
        const glm::u16vec3* indices = reinterpret_cast<glm::u16vec3*>(sbt_data->mesh_indices.data);
        vertex_indices = glm::uvec3(indices[triangle_index]);
    }

    // Vertex positions of selected triangle
    glm::vec3 P0 = sbt_data->mesh_positions[vertex_indices.x];
    glm::vec3 P1 = sbt_data->mesh_positions[vertex_indices.y];
    glm::vec3 P2 = sbt_data->mesh_positions[vertex_indices.z];

    // Compute local position
    glm::vec3 local_light_position = (1.0f-triangle_barys.x-triangle_barys.y)*P0 + triangle_barys.x*P1 + triangle_barys.y*P2;
    // Transform local position to world position
    glm::vec3 light_position = glm::vec3(sbt_data->local_to_world * glm::vec4(local_light_position, 1));

    // Compute local normal
    glm::vec3 local_light_normal = glm::cross(P1-P0, P2-P0);
    // Normals are transformed by (A^-1)^T instead of A
    glm::vec3 light_normal = glm::normalize(glm::transpose(glm::mat3(sbt_data->world_to_local)) * local_light_normal);


    // Assemble sampling result

    EmitterSamplingResult result;
    result.sampling_pdf = 0; // initialize with invalid sample

    // light source sampling
    result.direction_to_light = glm::normalize(light_position - si.position);
    result.distance_to_light = glm::length(light_position - si.position);
    result.normal_at_light = light_normal;

    float one_over_light_position_pdf = sbt_data->total_surface_area;
    float cos_theta_on_light = glm::abs(glm::dot(result.direction_to_light, light_normal));
    float one_over_light_direction_pdf = one_over_light_position_pdf * cos_theta_on_light / (result.distance_to_light * result.distance_to_light);

    result.radiance_weight_at_receiver = sbt_data->radiance * one_over_light_direction_pdf;

    // Probability of sampling this direction via light source sampling
    result.sampling_pdf = 1 / one_over_light_direction_pdf;

    return result;
}

extern "C" __device__ float __direct_callable__meshlight_evalLightSamplingPDF(const Interaction &si, const SurfaceInteraction &si_on_light)
{
    const MeshLightData *sbt_data = *reinterpret_cast<const MeshLightData **>(optixGetSbtDataPointer());
    // We can assume that outgoing ray dir actually intersects the light source.

    // Some useful quantities
    glm::vec3 light_normal = si_on_light.normal;
    glm::vec3 light_ray_dir = si_on_light.incoming_ray_dir; // glm::normalize(si_on_light.position - si.position);
    float light_ray_length = si_on_light.incoming_distance; // glm::length(si_on_light.position - si.position);

    // The probability of sampling any position on the surface of the mesh is the reciprocal of its surface area.
    float light_position_pdf = 1 / sbt_data->total_surface_area;

    // Probability of sampling this direction via light source sampling
    float cos_theta_on_light = glm::abs(glm::dot(light_ray_dir, light_normal));
    float light_direction_pdf = light_position_pdf * light_ray_length * light_ray_length / cos_theta_on_light;

    return light_direction_pdf;
}

extern "C" __device__ EmitterPhotonSamplingResult __direct_callable__meshlight_samplePhoton(PCG32 &rng)
{
    const MeshLightData *sbt_data = *reinterpret_cast<const MeshLightData **>(optixGetSbtDataPointer());

    EmitterPhotonSamplingResult result;
    result.sampling_pdf = 0; // invalid photon

    // Select the triangle to sample a direction from uniformly at random, proportional to its surface area
    uint32_t triangle_index = 0;
    // Sample the barycentric coordinates on the triangle uniformly.
    glm::vec2 triangle_barys = glm::vec2(0, 0);

    triangle_index = opg::binary_search(sbt_data->mesh_cdf, rng.next1d());

    triangle_barys = rng.next2d();
    // Mirror barys at diagonal line to cover a triangle instead of a square
    if (triangle_barys.x + triangle_barys.y > 1)
        triangle_barys = glm::vec2(1) - triangle_barys;


    // Compute the `light_position` using the triangle_index and the triangle_barys on the mesh:

    // Indices of triangle vertices in the mesh
    glm::uvec3 vertex_indices = glm::uvec3(0u);
    if (sbt_data->mesh_indices.elmt_byte_size == sizeof(glm::u32vec3))
    {
        // Indices stored as 32-bit unsigned integers
        const glm::u32vec3* indices = reinterpret_cast<glm::u32vec3*>(sbt_data->mesh_indices.data);
        vertex_indices = glm::uvec3(indices[triangle_index]);
    }
    else
    {
        // Indices stored as 16-bit unsigned integers
        const glm::u16vec3* indices = reinterpret_cast<glm::u16vec3*>(sbt_data->mesh_indices.data);
        vertex_indices = glm::uvec3(indices[triangle_index]);
    }

    // Vertex positions of selected triangle
    glm::vec3 P0 = sbt_data->mesh_positions[vertex_indices.x];
    glm::vec3 P1 = sbt_data->mesh_positions[vertex_indices.y];
    glm::vec3 P2 = sbt_data->mesh_positions[vertex_indices.z];

    // Compute local position
    glm::vec3 local_photon_position = (1.0f-triangle_barys.x-triangle_barys.y)*P0 + triangle_barys.x*P1 + triangle_barys.y*P2;
    // Transform local position to world position
    glm::vec3 photon_origin = glm::vec3(sbt_data->local_to_world * glm::vec4(local_photon_position, 1));
    float photon_origin_pdf = 1 / sbt_data->total_surface_area;

    // Compute local normal
    glm::vec3 local_light_normal = glm::cross(P1-P0, P2-P0);
    // If the surface is double sided, flip the local light normal half of the time!
    // The probability of being on either side is already captured in photon_origin_pdf since sbt_data->total_surface_area includes the double-sidedness
    if (sbt_data->double_sided && rng.next1d() < 0.5f)
        local_light_normal *= -1;
    // Normals are transformed by (A^-1)^T instead of A
    glm::vec3 light_normal = glm::normalize(glm::transpose(glm::mat3(sbt_data->world_to_local)) * local_light_normal);
    // Compute local frame around light normal
    glm::mat3 light_normal_frame = opg::compute_local_frame(light_normal);
    // Compute photon direction in local frame
    glm::vec3 local_photon_direction = warp_square_to_hemisphere_cosine(rng.next2d());
    // Transform photon direction to world space
    glm::vec3 photon_direction = light_normal_frame * local_photon_direction;
    float photon_direction_pdf = warp_square_to_hemisphere_cosine_pdf(local_photon_direction);


    float NdotL = glm::dot(light_normal, photon_direction);

    // Populate result
    result.position = photon_origin;
    result.direction = photon_direction;
    result.normal_at_light = light_normal;
    float one_over_photon_origin_pdf = sbt_data->total_surface_area;
    //float one_over_photon_dir_pdf = glm::pi<float>() / NdotL;
    // result.radiance = NdotL * sbt_data->radiance;
    // result.radiance_weight = result.radiance * one_over_photon_dir_pdf * one_over_photon_origin_pdf;
    result.radiance_weight = sbt_data->radiance * glm::pi<float>() * one_over_photon_origin_pdf;
    result.sampling_pdf = photon_origin_pdf * photon_direction_pdf;

    return result;
}
