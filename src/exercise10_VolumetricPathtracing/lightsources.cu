#include "hip/hip_runtime.h"
#include "lightsources.cuh"

#include "opg/scene/interface/emitter.cuh"
#include "opg/scene/utility/interaction.cuh"
#include "opg/hostdevice/binarysearch.h"
#include "opg/hostdevice/coordinates.h"

#include <optix.h>


extern "C" __device__ EmitterSamplingResult __direct_callable__pointlight_sampleLight(const Interaction &si, PCG32 &unused_rng)
{
    const PointLightData *sbt_data = *reinterpret_cast<const PointLightData **>(optixGetSbtDataPointer());

    glm::vec3 dir_to_light = sbt_data->position - si.position;

    EmitterSamplingResult result;
    result.radiance_weight_at_receiver = sbt_data->intensity / glm::dot(dir_to_light, dir_to_light);
    result.direction_to_light = glm::normalize(dir_to_light);
    result.distance_to_light = glm::length(dir_to_light);
    result.sampling_pdf = 1;
    return result;
}

extern "C" __device__ EmitterSamplingResult __direct_callable__directionallight_sampleLight(const Interaction &si, PCG32 &unused_rng)
{
    const DirectionalLightData *sbt_data = *reinterpret_cast<const DirectionalLightData **>(optixGetSbtDataPointer());

    glm::vec3 dir_to_light = sbt_data->direction;

    EmitterSamplingResult result;
    result.radiance_weight_at_receiver = sbt_data->irradiance_at_receiver;
    result.direction_to_light = glm::normalize(dir_to_light);
    result.distance_to_light = std::numeric_limits<float>::infinity();
    result.sampling_pdf = 1;
    return result;
}



__device__ glm::vec3 warp_square_to_spherical_cap_uniform(const glm::vec2 &uv, float cap_height)
{
    // See https://en.wikipedia.org/wiki/Spherical_cap

    float z = glm::lerp(1.0f-cap_height, 1.0f, uv.x);
    float phi = 2*M_PIf * uv.y;
    float r = glm::sqrt(1 - z*z);
    float x = r * glm::cos(phi);
    float y = r * glm::sin(phi);

    return glm::vec3(x, y, z);
}



extern "C" __device__ glm::vec3 __direct_callable__spherelight_evalLight(const SurfaceInteraction &si)
{
    const SphereLightData *sbt_data = *reinterpret_cast<const SphereLightData **>(optixGetSbtDataPointer());
    // We can assume that si is actually on the surface of the light source

    // The emitted radiance is constant across the light source
    return sbt_data->radiance;
}

extern "C" __device__ EmitterSamplingResult __direct_callable__spherelight_sampleLight(const Interaction &si, PCG32 &rng)
{
    const SphereLightData *sbt_data = *reinterpret_cast<const SphereLightData **>(optixGetSbtDataPointer());

    // Some useful quantities
    glm::vec3 light_center_dir = glm::normalize(sbt_data->position - si.position);
    float light_center_distance = glm::length(sbt_data->position - si.position);
    float cap_height = 1 - glm::sqrt(light_center_distance*light_center_distance - sbt_data->radius*sbt_data->radius) / light_center_distance;

    // A transformation matrix that rotates the z axis to the light_center_dir
    glm::mat3 local_frame = opg::compute_local_frame(light_center_dir);


    EmitterSamplingResult result;

    if (light_center_distance < sbt_data->radius)
    {
        result.radiance_weight_at_receiver = glm::vec3(0);
        result.sampling_pdf = 0;
        return result;
    }

    // Light source sampling
    glm::vec3 local_direction_to_light = warp_square_to_spherical_cap_uniform(rng.next2d(), cap_height);
    result.direction_to_light = local_frame * local_direction_to_light;

    float spherical_cap_area = 2*glm::pi<float>() * cap_height;

    // Radiance divided by sampling pdf
    result.radiance_weight_at_receiver = sbt_data->radiance * spherical_cap_area;

    // Probability of sampling this direction via light source sampling
    result.sampling_pdf = 1 / spherical_cap_area;


    // Compute light_ray_length for occlusion query
    {
        glm::vec3 O = si.position - sbt_data->position;
        glm::vec3 D = result.direction_to_light;

        float p = glm::dot(D, O) / glm::dot(D, D); // p/2 actually
        float q = (glm::dot(O, O) - sbt_data->radius * sbt_data->radius) / glm::dot(D, D);

        float k = glm::max(p*p - q, 0.0f);
        // Usually we would have to check for k < 0, but by construction the ray intersects the light source.

        // Assuming the surface element is outside of the sphere, the first intersection is what we want!
        result.distance_to_light = -p - glm::sqrt(k);
    }

    return result;
}

extern "C" __device__ float __direct_callable__spherelight_evalLightSamplingPDF(const Interaction &si, const SurfaceInteraction &si_on_light)
{
    const SphereLightData *sbt_data = *reinterpret_cast<const SphereLightData **>(optixGetSbtDataPointer());
    // We can assume that outgoing ray dir actually intersects the light source.

    // Some useful quantities
    float light_center_distance = glm::length(sbt_data->position - si.position);
    float cap_height = 1 - glm::sqrt(light_center_distance*light_center_distance - sbt_data->radius*sbt_data->radius) / light_center_distance;

    // Probability of sampling this direction via light source sampling
    float sampling_pdf = 1 / (2*glm::pi<float>() * cap_height);

    return sampling_pdf;
}


extern "C" __device__ glm::vec3 __direct_callable__meshlight_evalLight(const SurfaceInteraction &si)
{
    const MeshLightData *sbt_data = *reinterpret_cast<const MeshLightData **>(optixGetSbtDataPointer());
    // We can assume that si is actually on the surface of the light source

    // The emitted radiance is constant across the light source
    return sbt_data->radiance;
}

extern "C" __device__ EmitterSamplingResult __direct_callable__meshlight_sampleLight(const Interaction &si, PCG32 &rng)
{
    const MeshLightData *sbt_data = *reinterpret_cast<const MeshLightData **>(optixGetSbtDataPointer());

    // Select the triangle to sample a direction from uniformly at random, proportional to its surface area
    uint32_t triangle_index = 0;
    // Sample the barycentric coordinates on the triangle uniformly.
    glm::vec2 triangle_barys = glm::vec2(0, 0);

    triangle_index = opg::binary_search(sbt_data->mesh_cdf, rng.next1d());

    triangle_barys = rng.next2d();
    // Mirror barys at diagonal line to cover a triangle instead of a square
    if (triangle_barys.x + triangle_barys.y > 1)
        triangle_barys = glm::vec2(1) - triangle_barys;


    // Compute the `light_position` using the triangle_index and the triangle_barys on the mesh:

    // Indices of triangle vertices in the mesh
    glm::uvec3 vertex_indices = glm::uvec3(0u);
    if (sbt_data->mesh_indices.elmt_byte_size == sizeof(glm::u32vec3))
    {
        // Indices stored as 32-bit unsigned integers
        const glm::u32vec3* indices = reinterpret_cast<glm::u32vec3*>(sbt_data->mesh_indices.data);
        vertex_indices = glm::uvec3(indices[triangle_index]);
    }
    else
    {
        // Indices stored as 16-bit unsigned integers
        const glm::u16vec3* indices = reinterpret_cast<glm::u16vec3*>(sbt_data->mesh_indices.data);
        vertex_indices = glm::uvec3(indices[triangle_index]);
    }

    // Vertex positions of selected triangle
    glm::vec3 P0 = sbt_data->mesh_positions[vertex_indices.x];
    glm::vec3 P1 = sbt_data->mesh_positions[vertex_indices.y];
    glm::vec3 P2 = sbt_data->mesh_positions[vertex_indices.z];

    // Compute local position
    glm::vec3 local_light_position = (1.0f-triangle_barys.x-triangle_barys.y)*P0 + triangle_barys.x*P1 + triangle_barys.y*P2;
    // Transform local position to world position
    glm::vec3 light_position = glm::vec3(sbt_data->local_to_world * glm::vec4(local_light_position, 1));

    // Compute local normal
    glm::vec3 local_light_normal = glm::cross(P1-P0, P2-P0);
    // Normals are transformed by (A^-1)^T instead of A
    glm::vec3 light_normal = glm::normalize(glm::transpose(glm::mat3(sbt_data->world_to_local)) * local_light_normal);


    // Assemble sampling result

    EmitterSamplingResult result;
    result.sampling_pdf = 0; // initialize with invalid sample

    // light source sampling
    result.direction_to_light = glm::normalize(light_position - si.position);
    result.distance_to_light = glm::length(light_position - si.position);


    // p(A)dA = p(W)dW => p(W) = p(A)*|dA/dW|
    // |dA/dW| = r^2/cos(theta)
    float one_over_light_position_pdf = sbt_data->total_surface_area;
    float cos_theta_on_light = glm::abs(glm::dot(result.direction_to_light, light_normal));
    float one_over_light_direction_pdf = one_over_light_position_pdf * cos_theta_on_light / (result.distance_to_light * result.distance_to_light);

    result.radiance_weight_at_receiver = sbt_data->radiance * one_over_light_direction_pdf;

    // Probability of sampling this direction via light source sampling
    result.sampling_pdf = 1 / one_over_light_direction_pdf;

    return result;
}

extern "C" __device__ float __direct_callable__meshlight_evalLightSamplingPDF(const Interaction &si, const SurfaceInteraction &si_on_light)
{
    const MeshLightData *sbt_data = *reinterpret_cast<const MeshLightData **>(optixGetSbtDataPointer());
    // We can assume that outgoing ray dir actually intersects the light source.

    // Some useful quantities
    glm::vec3 light_normal = si_on_light.normal;
    glm::vec3 light_ray_dir = si_on_light.incoming_ray_dir; // glm::normalize(si_on_light.position - si.position);
    float light_ray_length = si_on_light.incoming_distance; // glm::length(si_on_light.position - si.position);

    // The probability of sampling any position on the surface of the mesh is the reciprocal of its surface area.
    float light_position_pdf = 1 / sbt_data->total_surface_area;

    // Probability of sampling this direction via light source sampling
    float cos_theta_on_light = glm::abs(glm::dot(light_ray_dir, light_normal));
    float light_direction_pdf = light_position_pdf * light_ray_length * light_ray_length / cos_theta_on_light;

    return light_direction_pdf;
}
